#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>

#define SIZE 9

__device__ bool is_valid_gpu(char board[SIZE][SIZE], int row, int col, char num) {
    //row
    for (int i = 0; i < SIZE; i++) {
        if (board[row][i] == num) {
            return false;
        }
    }
    //col
    for (int i = 0; i < SIZE; i++) {
        if (board[i][col] == num) {
            return false;
        }
    }
    //subcuadro
    int start_row = 3 * (row / 3);
    int start_col = 3 * (col / 3);
    for (int i = start_row; i < start_row + 3; i++) {
        for (int j = start_col; j < start_col + 3; j++) {
            if (board[i][j] == num) {
                return false;
            }
        }
    }
    return true;
}

__device__ bool solve_sudoku_recursive_gpu(char board[SIZE][SIZE], int row, int col) {
    while (row < SIZE && board[row][col] != '.') {
        col++;
        if (col == SIZE) {
            col = 0;
            row++;
        }
    }

    //resuelto
    if (row == SIZE) {
        return true;
    }

    for (char num = '1'; num <= '9'; num++) {
        if (is_valid_gpu(board, row, col, num)) {

            board[row][col] = num;

            //recursion
            if (solve_sudoku_recursive_gpu(board, row, col)) {
                return true;
            }

            //probar otro numero
            board[row][col] = '.';
        }
    }

    //no se pudo :c
    return false;
}

__global__ void solve_sudoku_kernel(char* dev_board, bool* dev_result) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    char board[SIZE][SIZE];
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            board[i][j] = dev_board[index * SIZE * SIZE + i * SIZE + j];
        }
    }
    dev_result[index] = solve_sudoku_recursive_gpu(board, 0, 0);
}

bool solve_sudoku_gpu(char board[SIZE][SIZE]) {
    std::vector<char> board_vec;
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            board_vec.push_back(board[i][j]);
        }
    }

    char* dev_board;
    bool* dev_result;
    hipMalloc((void**)&dev_board, SIZE * SIZE * sizeof(char));
    hipMalloc((void**)&dev_result, SIZE * sizeof(bool));

    hipMemcpy(dev_board, board_vec.data(), SIZE * SIZE * sizeof(char), hipMemcpyHostToDevice);

    solve_sudoku_kernel << <SIZE, 1 >> > (dev_board, dev_result);

    bool result[SIZE];
    hipMemcpy(result, dev_result, SIZE * sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(dev_board);
    hipFree(dev_result);

    for (int i = 0; i < SIZE; i++) {
        if (result[i]) {
            return true;
        }
    }
    return false;
}

int main() {
    // Tablero de ejemplo
    char board[SIZE][SIZE] = {
        {'5', '3', '.', '.', '7', '.', '.', '.', '.'},
        {'6', '.', '.', '1', '9', '5', '.', '.', '.'},
        {'.', '9', '8', '.', '.', '.', '.', '6', '.'},
        {'8', '.', '.', '.', '6', '.', '.', '.', '3'},
        {'4', '.', '.', '8', '.', '3', '.', '.', '1'},
        {'7', '.', '.', '.', '2', '.', '.', '.', '6'},
        {'.', '6', '.', '.', '.', '.', '2', '8', '.'},
        {'.', '.', '.', '4', '1', '9', '.', '.', '5'},
        {'.', '.', '.', '.', '8', '.', '.', '7', '9'}
    };

    std::cout << "tablero original:" << std::endl;
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            std::cout << board[i][j] << " ";
        }
        std::cout << std::endl;
    }

    if (solve_sudoku_gpu(board)) {
        std::cout << "\nsolucion:" << std::endl;
        for (int i = 0; i < SIZE; i++) {
            for (int j = 0; j < SIZE; j++) {
                std::cout << board[i][j] << " ";
            }
            std::cout << std::endl;
        }
    }
    else {
        std::cout << "\nno se pudo resolver el sudoku :c" << std::endl;
    }

    return 0;
}
//no lo resuelve :cc lo deja igual
